
#include <hip/hip_runtime.h>
/**
 * 
 * @author Lokesh Agrawal
 * @version 4-NOV-2016
 *
 */
#define NT 1024

// Structure for a 1-D vector of points.
typedef struct
   {
   double x;
   double y;
   double z;
   }
   vector_points;

// Structure for a 1-D vector of result.   
typedef struct
   {
   int x;
   int y;
   double distance;
   }
   vector_result;

// Thread rank that found the best-of-best solution.
__device__ int bobRank;

// For shared memory reduction of threads for a particular pair.
__shared__ double shrRank[NT];

extern "C" __global__ void compute(vector_points *pointsList, 
vector_result *result, int N){
		
   	int k,oldRank, newRank, blockId;
	vector_points point1, point2;
	double distance1 = 0;
	double distance2 = 0;
	
	// Index of this thread within 
	int localThreadRank = threadIdx.x;
		
	  //blockId stores block id of a block in a gpu
	  blockId = blockIdx.x;
	  result[blockId].x = 0;
	  result[blockId].y = 0;
	  result[blockId].distance = 0;
	  point1 = pointsList[blockId];
   	  for (k = blockId+1; k < N; k++)
      {      
      		shrRank[localThreadRank] = 0;       	
      		point2 = pointsList[k];
      		
      		//Each thread calculates the distance of points for a particular pair
      		//of points.   		
      		for(int currentPoint=localThreadRank;currentPoint<N; currentPoint+=NT){
      			if(currentPoint==k || currentPoint==blockId)	continue;
      			distance1 = abs(pointsList[currentPoint].x-point1.x)+abs(pointsList[currentPoint].y-point1.y)+abs(pointsList[currentPoint].z-point1.z);
      			distance2 = abs(pointsList[currentPoint].x-point2.x)+abs(pointsList[currentPoint].y-point2.y)+abs(pointsList[currentPoint].z-point2.z);
      			
      			//The minimum distance out of two is added.
      			if(distance1<distance2)		shrRank[localThreadRank] = shrRank[localThreadRank]+distance1;
      			else	shrRank[localThreadRank] = 	shrRank[localThreadRank]+distance2;
     		}
  			
  			//Reduction of threads happens below to add distances of all threads
  			// and final result is stored at 0 index
  			__syncthreads();
    		 for (int p = NT/2; p > 0; p >>= 1)
      		{
      			if (localThreadRank < p && localThreadRank+p < NT){
         				shrRank[localThreadRank] = shrRank[localThreadRank] + shrRank[localThreadRank+p];
         		}
         		//Syncthread is must at this place to let the reduction of all
         		// threads complete
     			__syncthreads();
      		}		
  			
  			//if this block has no result till now then we copy the obtained
  			//result. Otherwise, result with min distance is copied.
  			if(result[blockId].distance==0){
  				result[blockId].distance = shrRank[0];
  				result[blockId].x = blockId;
  				result[blockId].y = k;
  			}
  			else if(shrRank[0]<result[blockId].distance){
  				result[blockId].distance = shrRank[0];
  				result[blockId].x = blockId;
  				result[blockId].y = k;
  			}
  				
      }

   // Global memory reduction to determine thread rank with best-of-best
   // solution across all blocks.
   if (localThreadRank == 0 && result[blockId].distance!=0)
      do
         {
         oldRank = bobRank;
         newRank =
            oldRank == -1 ||
            result[blockId].distance < result[oldRank].distance ?
               blockId : oldRank;
         }
      while (atomicCAS (&bobRank, oldRank, newRank) != oldRank);
   }